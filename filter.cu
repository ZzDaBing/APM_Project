#include "hip/hip_runtime.h"
#include <iostream>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include "filter.h"

using namespace std;

// Saturation Filter
__global__ void saturation(unsigned int* d_img, unsigned int* d_tmp, int width, int height)
{
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
  
  if(idy < height && idx < width){
    int ida = ((idy * width) + idx) * 3;
    d_img[ida + 0] = 255;
    d_img[ida + 1] = d_tmp[ida + 1];
    d_img[ida + 2] = d_tmp[ida + 2];
  }
}

// Symetry Filter
__global__ void symetry(unsigned int* d_img, unsigned int* d_tmp, int width, int height)
{
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  int idy = (blockIdx.y * blockDim.y) + threadIdx.y;

  if(idy < height && idx < width){
    int ida = ((idy * width) + idx) * 3;
    int idinvers = ((width * height) - ((idy * width) + idx)) * 3;
    d_img[ida + 0] = d_tmp[idinvers];
    d_img[ida + 1] = d_tmp[idinvers + 1];
    d_img[ida + 2] = d_tmp[idinvers + 2];
  }
}

// Blur definition
__global__ void blur(unsigned int* d_img, unsigned int* d_tmp, int width, int height)
{
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  int idy = (blockIdx.y * blockDim.y) + threadIdx.y;

  if(idy < height && idx < width){
    int ida = ((idy * width) + idx) * 3;

    int avg_red = d_tmp[ida + 0];
    int avg_green = d_tmp[ida + 1];
    int avg_blue = d_tmp[ida + 2];

    //TOP BORDER
    if(idx < width && idy == 0){
      //Top-left corner
      if(ida == 0){ 
        avg_red += d_tmp[3] + d_tmp[(width * 3)];
        avg_green += d_tmp[4] + d_tmp[(width * 3) + 1];
        avg_blue += d_tmp[5] + d_tmp[(width * 3) + 2];

        avg_red /= 3;
        avg_green /= 3;
        avg_blue /= 3;
      }
      else{
        //Top-right corner
        if(ida == width - 1){ 
          avg_red += d_tmp[ida - 3] + d_tmp[ida + (width * 3)];
          avg_green += d_tmp[ida - 2] + d_tmp[ida + (width * 3) + 1];
          avg_blue += d_tmp[ida - 1] + d_tmp[ida + (width * 3) + 2];

          avg_red /= 3;
          avg_green /= 3;
          avg_blue /= 3;
        }
        else{
          avg_red += d_tmp[ida - 3] + d_tmp[ida + 3] + d_tmp[ida + (width * 3)];
          avg_green += d_tmp[ida - 2] + d_tmp[ida + 4] + d_tmp[ida + (width * 3) + 1];
          avg_blue += d_tmp[ida - 1] + d_tmp[ida + 5] + d_tmp[ida + (width * 3) + 2];

          avg_red /= 4;
          avg_green /= 4;
          avg_blue /= 4;
        }
      }
    }

    //BOTTOM BORDER
    if(idy == (height - 1)){
      //Bottom-left corner
      if(idx == 0){
        avg_red += d_tmp[ida + 3] + d_tmp[(ida - width * 3)];
        avg_green += d_tmp[ida + 4] + d_tmp[(ida - width * 3) + 1];
        avg_blue += d_tmp[ida + 5] + d_tmp[(ida - width * 3) + 2];

        avg_red /= 3;
        avg_green /= 3;
        avg_blue /= 3;
      }
      else{
        //Bottom-right corner
        if(idx == (width - 1)){
          avg_red += d_tmp[ida - 3] + d_tmp[ida - (width * 3)];
          avg_green += d_tmp[ida - 2] + d_tmp[ida - (width * 3) + 1];
          avg_blue += d_tmp[ida - 1] + d_tmp[ida - (width * 3) + 2];

          avg_red /= 3;
          avg_green /= 3;
          avg_blue /= 3;
        }
        else{
          avg_red += d_tmp[ida - 3] + d_tmp[ida + 3] + d_tmp[ida - (width * 3)];
          avg_green += d_tmp[ida - 2] + d_tmp[ida + 4] + d_tmp[ida - (width * 3) + 1];
          avg_blue += d_tmp[ida - 1] + d_tmp[ida + 5] + d_tmp[ida - (width * 3) + 2];

          avg_red /= 4;
          avg_green /= 4;
          avg_blue /= 4;
        }
      }
    }

    //LEFT BORDER (without corners)
    if( idx == 0 && idy != 0 && idy != height - 1 ){
      avg_red += d_tmp[(ida - width * 3)] + d_tmp[ida + 3] + d_tmp[(ida + width * 3)];
      avg_green += d_tmp[(ida - width * 3) + 1] + d_tmp[ida + 4] + d_tmp[(ida + width * 3) + 1];
      avg_blue += d_tmp[(ida - width * 3) + 2] + d_tmp[ida + 5] + d_tmp[(ida + width * 3) + 2];

      avg_red /= 4;
      avg_green /= 4;
      avg_blue /= 4;
    }

    //RIGHT BORDER (without corners)
    if( idx == width - 1 && idy != 0 && idy != height - 1 ){
      avg_red += d_tmp[(ida - width * 3)] + d_tmp[ida - 3] + d_tmp[(ida + width * 3)];
      avg_green += d_tmp[(ida - width * 3) + 1] + d_tmp[ida - 2] + d_tmp[(ida + width * 3) + 1];
      avg_blue += d_tmp[(ida - width * 3) + 2] + d_tmp[ida - 1] + d_tmp[(ida + width * 3) + 2];

      avg_red /= 4;
      avg_green /= 4;
      avg_blue /= 4;
    }

    //
    if( (idx > 0) && (idx < (width - 1)) && (idy > 0) && (idy < (height - 1)) ){
      avg_red += d_tmp[(ida - width * 3)] + d_tmp[ida - 3]  + d_tmp[ida + 3] + d_tmp[(ida + width * 3)];
      avg_green += d_tmp[(ida - width * 3) + 1] + d_tmp[ida - 2]  + d_tmp[ida + 4] + d_tmp[(ida + width * 3) + 1];
      avg_blue += d_tmp[(ida - width * 3) + 2] + d_tmp[ida - 1]  + d_tmp[ida + 5] + d_tmp[(ida + width * 3) + 2];

      avg_red /= 5;
      avg_green /= 5;
      avg_blue /= 5;
    }

    //Update pixel color
    d_img[ida + 0] = avg_red;
    d_img[ida + 1] = avg_green;
    d_img[ida + 2] = avg_blue;

  }
}

// Grayscale Filter
__global__ void grayscale(unsigned int* d_img, unsigned int* d_tmp, int width, int height)
{
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  int idy = (blockIdx.y * blockDim.y) + threadIdx.y;

  if(idy < height && idx < width){
    int ida = ((idy * width) + idx) * 3;
    double val = (0.299*d_tmp[ida + 0]) + (0.587*d_tmp[ida + 1]) + (0.114*d_tmp[ida + 2]);
    d_img[ida + 0] = (int)val;
    d_img[ida + 1] = (int)val;
    d_img[ida + 2] = (int)val;
  }
}

// Sobel Filter
__global__ void sobel(unsigned int* d_img, unsigned int* d_tmp, int width, int height) {
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  int idy = (blockIdx.y * blockDim.y) + threadIdx.y;

  if(idy < height && idx < width){
    int ida_1 = (((idy-1) * width) + idx) * 3;
    int ida_2 = ((idy     * width) + idx+1) * 3;
    int ida_3 = (((idy+1) * width) + idx+1) * 3;
    int ida_4 = (((idy-1) * width) + idx)   * 3;
    int ida_5 = ((idy     * width) + idx)   * 3;
    int ida_6 = (((idy+1) * width) + idx)   * 3;
    int ida_7 = (((idy-1) * width) + idx-1) * 3;
    int ida_8 = ((idy     * width) + idx-1) * 3;
    int ida_9 = (((idy+1) * width) + idx-1) * 3;

    int Gx = 0, Gy = 0;

    if (idy < height-1 && idy > 0 && idx < width-1 && idx > 0){
        Gx = -1 * d_tmp[ida_7] + d_tmp[ida_1]
            - 2 * d_tmp[ida_8] + 2 * d_tmp[ida_2]
            - d_tmp[ida_9] + d_tmp[ida_3];
        Gy = -1 * d_tmp[ida_7] - 2 * d_tmp[ida_4]
            - d_tmp[ida_1] + d_tmp[ida_9]
            + 2 * d_tmp[ida_6] + d_tmp[ida_3];
    }

    int sum = Gx * Gx + Gy * Gy;
    int res = sqrt((float)sum);
    d_img[ida_5] = res;
    d_img[ida_5 + 1] = res;
    d_img[ida_5 + 2] = res;
  }
}

// Negative Filter
__global__ void negative(unsigned int* d_img, unsigned int* d_tmp, int width, int height) {
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  int idy = (blockIdx.y * blockDim.y) + threadIdx.y;

  if(idy < height && idx < width){
    int ida = ((idy * width) + idx) * 3;
    d_img[ida + 0] = 255 - d_tmp[ida];
    d_img[ida + 1] = 255 - d_tmp[ida + 1];
    d_img[ida + 2] = 255 - d_tmp[ida + 2];
  }
}

// Only-one-color Filter
__global__ void only_blue(unsigned int* d_img, unsigned int* d_tmp, int width, int height) {
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  int idy = (blockIdx.y * blockDim.y) + threadIdx.y;

  if(idy < height && idx < width){
    int ida = ((idy * width) + idx) * 3;
    d_img[ida + 0] = 0;
    d_img[ida + 1] = 0;
    d_img[ida + 2] = d_tmp[ida + 2];
  }
}

// Rotate 90
__global__ void rotate90(unsigned int* d_img, unsigned int* d_tmp, int width, int height) {
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  int idy = (blockIdx.y * blockDim.y) + threadIdx.y;

  if(idy < height && idx < width){
    int ida = ((idy * width) + idx) * 3;
    int ida_2 = ((idx * width) + (height-idy-1)) * 3;
    d_img[ida_2] = d_tmp[ida];
    d_img[ida_2 + 1] = d_tmp[ida + 1];
    d_img[ida_2 + 2] = d_tmp[ida + 2];
  }
}

// Resize Filter
__global__ void resize(unsigned int* d_img, unsigned int* d_tmp, int width, int height, int new_width, int new_height)
{
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
  int ida = (idy * width + idx) * 3;

  if(idy < height && idx < width) {
    d_img[ida] = 0;
    d_img[ida + 1] = 0;
    d_img[ida + 2] = 0;
  }

  if (idx < new_width && idy < new_height) {
    double scale_width = (double)new_width / (double)width;
    double scale_height = (double)new_height / (double)height;
    int idx_2 = (int)((double)idx / scale_width);
    int idy_2 = (int)((double)idy / scale_height);

    int ida_2 = (idy_2 * width + idx_2) * 3;

    d_img[ida] = d_tmp[ida_2];
    d_img[ida + 1] = d_tmp[ida_2 + 1];
    d_img[ida + 2] = d_tmp[ida_2 + 2];
  }
}

// Popart filter
__global__ void photomaton(unsigned int* d_img, unsigned int* d_tmp, int width, int height)
{
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  int idy = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (idy < height && idx < width){
    int ida = ((idy * width) + idx) * 3;

    int x,y;
    int w_d = (int)width / 2;
    int h_d = (int)height / 2;

    x = idx;
    y = idy;

    if(idx % 2 == 0) x = (int)(idx / 2);
    else{ x = (int)( ((idx - 1) / 2) + w_d );}
    if(idy % 2 == 0) y = (int)(idy / 2);
    else{ y = (int)( ((idy - 1) / 2) + h_d );}

    int idb = ((y * width) + x) * 3;
    
    d_img[idb + 0] = d_tmp[ida + 0];
    d_img[idb + 1] = d_tmp[ida + 1];
    d_img[idb + 2] = d_tmp[ida + 2];
  }
}

void popart(dim3 nbBlocks, dim3 nbThreadsPerBlock, unsigned int *img, unsigned int *d_img, unsigned int *d_tmp, int width, int height)
{
  resize<<<nbBlocks, nbThreadsPerBlock>>>(d_img, d_tmp, width, height, width/2, height/2);
  hipMemcpy(img, d_img, 3 * width * height * sizeof(unsigned int), hipMemcpyDeviceToHost);

  //Streams
  hipStream_t streams[4];
  for (int i = 0; i < 4; ++i)
    hipStreamCreate(&streams[i]);

  //Small images
  unsigned int *topl, *topr, *botl, *botr, *d_topl, *d_topr, *d_botl, *d_botr, *d_tmptl, *d_tmptr, *d_tmpbl, *d_tmpbr;
  topl = (unsigned int*) malloc(sizeof(unsigned int) * 3 * ((width * height) / 2));
  topr = (unsigned int*) malloc(sizeof(unsigned int) * 3 * ((width * height) / 2));
  botl = (unsigned int*) malloc(sizeof(unsigned int) * 3 * ((width * height) / 2));
  botr = (unsigned int*) malloc(sizeof(unsigned int) * 3 * ((width * height) / 2));
  hipMalloc(&d_topl, sizeof(unsigned int) * 3 * ((width * height) / 2));
  hipMalloc(&d_topr, sizeof(unsigned int) * 3 * ((width * height) / 2)); 
  hipMalloc(&d_botl, sizeof(unsigned int) * 3 * ((width * height) / 2)); 
  hipMalloc(&d_botr, sizeof(unsigned int) * 3 * ((width * height) / 2));
  hipMalloc(&d_tmptl, sizeof(unsigned int) * 3 * ((width * height) / 2));
  hipMalloc(&d_tmptr, sizeof(unsigned int) * 3 * ((width * height) / 2));
  hipMalloc(&d_tmpbl, sizeof(unsigned int) * 3 * ((width * height) / 2));
  hipMalloc(&d_tmpbr, sizeof(unsigned int) * 3 * ((width * height) / 2));

  //Splits
  for (int i = 0; i < height / 2; ++i){
    for(int j = 0; j < width / 2; ++j){
      botl[(i * (width / 2) + j) * 3 + 0] = img[(i * width + j) * 3 + 0];
      botl[(i * (width / 2) + j) * 3 + 1] = img[(i * width + j) * 3 + 1];
      botl[(i * (width / 2) + j) * 3 + 2] = img[(i * width + j) * 3 + 2];

      botr[(i * (width / 2) + j) * 3 + 0] = img[(i * width + j) * 3 + 0];
      botr[(i * (width / 2) + j) * 3 + 1] = img[(i * width + j) * 3 + 1];
      botr[(i * (width / 2) + j) * 3 + 2] = img[(i * width + j) * 3 + 2];

      topl[(i * (width / 2) + j) * 3 + 0] = img[(i * width + j) * 3 + 0];
      topl[(i * (width / 2) + j) * 3 + 1] = img[(i * width + j) * 3 + 1];
      topl[(i * (width / 2) + j) * 3 + 2] = img[(i * width + j) * 3 + 2];

      topr[(i * (width / 2) + j) * 3 + 0] = img[(i * width + j) * 3 + 0];
      topr[(i * (width / 2) + j) * 3 + 1] = img[(i * width + j) * 3 + 1];
      topr[(i * (width / 2) + j) * 3 + 2] = img[(i * width + j) * 3 + 2];
    }
  }

  //Copy 
  hipMemcpyAsync(d_topl, topl, 3 * ((width * height) / 2) * sizeof(unsigned int), hipMemcpyHostToDevice, streams[0]);
  hipMemcpyAsync(d_topr, topr, 3 * ((width * height) / 2) * sizeof(unsigned int), hipMemcpyHostToDevice, streams[1]);
  hipMemcpyAsync(d_botl, botl, 3 * ((width * height) / 2) * sizeof(unsigned int), hipMemcpyHostToDevice, streams[2]);
  hipMemcpyAsync(d_botr, botr, 3 * ((width * height) / 2) * sizeof(unsigned int), hipMemcpyHostToDevice, streams[3]);
  hipMemcpyAsync(d_tmptl, d_topl, 3 * ((width * height) / 2) * sizeof(unsigned int), hipMemcpyDeviceToDevice, streams[0]);
  hipMemcpyAsync(d_tmptr, d_topr, 3 * ((width * height) / 2) * sizeof(unsigned int), hipMemcpyDeviceToDevice, streams[1]);
  hipMemcpyAsync(d_tmpbl, d_botl, 3 * ((width * height) / 2) * sizeof(unsigned int), hipMemcpyDeviceToDevice, streams[2]);
  hipMemcpyAsync(d_tmpbr, d_botr, 3 * ((width * height) / 2) * sizeof(unsigned int), hipMemcpyDeviceToDevice, streams[3]);
  //hipDeviceSynchronize();

  //Default filters applied
  only_blue<<<nbBlocks, nbThreadsPerBlock, 0, streams[0]>>>(d_topl, d_tmptl, height / 2, width / 2); //TOP LEFT
  negative<<<nbBlocks, nbThreadsPerBlock, 0, streams[1]>>>(d_topr, d_tmptr, height / 2, width / 2); //TOP RIGHT
  saturation<<<nbBlocks, nbThreadsPerBlock, 0, streams[2]>>>(d_botl, d_tmpbl, height / 2, width / 2);  //BOT LEFT
  symetry<<<nbBlocks, nbThreadsPerBlock, 0, streams[3]>>>(d_botr, d_tmpbr, height / 2, width / 2);  //BOT RIGHT

  hipMemcpyAsync(topl, d_topl, 3 * ((width * height) / 2) * sizeof(unsigned int), hipMemcpyDeviceToHost, streams[0]);
  hipMemcpyAsync(topr, d_topr, 3 * ((width * height) / 2) * sizeof(unsigned int), hipMemcpyDeviceToHost, streams[1]);
  hipMemcpyAsync(botl, d_botl, 3 * ((width * height) / 2) * sizeof(unsigned int), hipMemcpyDeviceToHost, streams[2]);
  hipMemcpyAsync(botr, d_botr, 3 * ((width * height) / 2) * sizeof(unsigned int), hipMemcpyDeviceToHost, streams[3]);
  hipDeviceSynchronize();

  //Regroups
  for (int i = 0; i < height / 2; ++i){
    for(int j = 0; j < width / 2; ++j){
      img[(i * width + j) * 3 + 0] = botl[(i * (width / 2) + j) * 3 + 0];
      img[(i * width + j) * 3 + 1] = botl[(i * (width / 2) + j) * 3 + 1];
      img[(i * width + j) * 3 + 2] = botl[(i * (width / 2) + j) * 3 + 2];

      img[(i * width + (width / 2) + j) * 3 + 0] = botr[(i * (width / 2) + j) * 3 + 0];
      img[(i * width + (width / 2) + j) * 3 + 1] = botr[(i * (width / 2) + j) * 3 + 1];
      img[(i * width + (width / 2) + j) * 3 + 2] = botr[(i * (width / 2) + j) * 3 + 2];

      img[((i + (height / 2)) * width + j) * 3 + 0] = topl[(i * (width / 2) + j) * 3 + 0];
      img[((i + (height / 2)) * width + j) * 3 + 1] = topl[(i * (width / 2) + j) * 3 + 1];
      img[((i + (height / 2)) * width + j) * 3 + 2] = topl[(i * (width / 2) + j) * 3 + 2];

      img[((i + (height / 2)) * width + (width / 2) + j) * 3 + 0] = topr[(i * (width / 2) + j) * 3 + 0];
      img[((i + (height / 2)) * width + (width / 2) + j) * 3 + 1] = topr[(i * (width / 2) + j) * 3 + 1];
      img[((i + (height / 2)) * width + (width / 2) + j) * 3 + 2] = topr[(i * (width / 2) + j) * 3 + 2];
    }
  }

  //Free memory

  for (int i = 0; i < 4; ++i)
    hipStreamDestroy(streams[i]);

  hipFree(d_topl);
  hipFree(d_topr);
  hipFree(d_botl);
  hipFree(d_botr);
  free(topl);
  free(topr);
  free(botl);
  free(botr);
}
